#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "CycleTimer.h"
#include <chrono>
#include <iostream>
#include <random>
#include <cmath>


#define BATCH_SIZE 100000
#define THREADS_PER_BLOCK 256

// CUDA Kernel for euclidean distance
__device__ float square_diff(float x, float y) {
    return (x - y) * (x - y);
}

__global__ void euclidean_distance_cuda(float* z, float *x, float *y, int d) {
    int index = blockIdx.y + THREADS_PER_BLOCK * threadIdx.x;
    float sqr_diff = 0.0;
    if (index < d) {
        sqr_diff = square_diff(x[index], y[index]);
        atomicAdd(z, sqr_diff);
    }
}

// Host function for euclidean distance using CUDA
void euclidean_distance(float* z, float *x, float *y, int d) {
    dim3 blocksPerGrid(1, d / THREADS_PER_BLOCK);
    dim3 threadsPerBlock(THREADS_PER_BLOCK, 1);
    euclidean_distance_cuda<<<blocksPerGrid, threadsPerBlock>>>(z, x, y, d);
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        fprintf(stderr, "Usage: %s [dimension]\n", argv[0]);
        exit(-1);
    }

    int d = atoi(argv[1]);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(-1.0, 1.0);

    float** x = (float**)malloc(BATCH_SIZE * sizeof(float*));
    float** y = (float**)malloc(BATCH_SIZE * sizeof(float*));
    float* z = (float*)malloc(BATCH_SIZE * sizeof(float));
    for(int i = 0; i < BATCH_SIZE; i++) {
        x[i] = (float*)malloc(d * sizeof(float));
        y[i] = (float*)malloc(d * sizeof(float));
    }

    for (int j = 0; j < BATCH_SIZE; j++) {
        for (int i = 0; i < d; ++i) {
            x[j][i] = dis(gen);
            y[j][i] = dis(gen);
        }
    }
    float *z_dev;
    float *x_dev;
    float *y_dev;
    double startTime = CycleTimer::currentSeconds();
    hipMalloc((void**)&z_dev, BATCH_SIZE * sizeof(float));
    hipMalloc((void**)&x_dev, d * sizeof(float));
    hipMalloc((void**)&y_dev, d * sizeof(float));
    double accTime = 0.0;
    double s_in = 0.0;
    double e_in = 0.0;
    for (int i = 0; i < BATCH_SIZE; i++) {
        hipMemcpy(x_dev, x[i], d * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(y_dev, y[i], d * sizeof(float), hipMemcpyHostToDevice);
        s_in = CycleTimer::currentSeconds();
        euclidean_distance(&z_dev[i], x_dev, y_dev, d);
        e_in = CycleTimer::currentSeconds();
        accTime += (e_in - s_in);
        //printf("z: %.4f\n", z[i]);
    }
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    hipMemcpy(z, z_dev, BATCH_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    printf("CUDA version took: %.4f seconds.\n", endTime - startTime);
    printf("CUDA kernels took: %.4f seconds for %d iterations.\n", accTime, BATCH_SIZE);
    double sum = 0.0;
    for (int i = 0; i < BATCH_SIZE; i++) {
        sum += z[i];
    }
    printf("sum: %.4f\n", sum);

    return 0;
}
